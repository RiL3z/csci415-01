#include "hip/hip_runtime.h"
//
// Assignment 1: ParallelSine
// CSCI 415: Networking and Parallel Computation
// Spring 2017
// Name(s): Kelan Riley
// Sine implementation derived from slides here: http://15418.courses.cs.cmu.edu/spring2016/lecture/basicarch


// standard imports
#include <stdio.h>
#include <math.h>
#include <iomanip>
#include <iostream>
#include <string>
#include <sys/time.h>

// problem size (vector length) N
// remember that a vector is just a series of values that we'd like to refer to
// as one thing, so we can refer to the whole series by just saying the word
// vector
static const int N = 134215680;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// need a better understanding of this algorithm for computing sine
// kernel function (CPU - Do not modify)
void sine_serial(float *input, float *output)
{
  // loop counter
  int i;

  // iterate as many times as there are numbers to work on
  for (i=0; i<N; i++) {
      // fetch ith number in the input array
      float value = input[i]; 
      // multiply the number by 3 initially
      float numer = input[i] * input[i] * input[i]; 
      int denom = 6; // 3! 
      int sign = -1; 
      // this loops TERMS number of times
      for (int j=1; j<=TERMS;j++) 
      { 
         value += sign * numer / denom; 
         numer *= input[i] * input[i]; 
         denom *= (2*j+2) * (2*j+3); 
         sign *= -1; 
      } 
      output[i] = value; 
    }
}


// kernel function (CUDA device)
// TODO: Implement your graphics kernel here. See assignment instructions for method information
// need to tell cuda that this is a kernel to run... need special syntax here...
//__global__ is the syntax for doing that, the below code will run on threads executing in the GPU
__global__ void sine_parallel(float *input, float *output) {
  // the thread id of the current thread that is running this kernel
  // threadIdx is a dim3 structure with x, y, and z fields (up to three dimensions)
  // make sure to compute the idx as the block number offset by the thread id in the block!
  int idx = blockIdx.x * 1024 + threadIdx.x;
  // fetch ith number in the input array
  float value = input[idx]; 
  // multiply the number by 3 initially
  float numer = input[idx] * input[idx] * input[idx]; 
  int denom = 6; // 3! 
  int sign = -1; 
  // this loops TERMS number of times
  for (int j=1; j<=TERMS;j++) 
  { 
    value += sign * numer / denom; 
    numer *= input[idx] * input[idx]; 
    denom *= (2*j+2) * (2*j+3); 
    sign *= -1; 
  }
  // write out the result into the output array 
  output[idx] = value;   
}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

// END: timing and error checking routines (do not modify)



int main (int argc, char **argv)
{
  // first I'm going to save the total number of bytes this array takes up in a variable
  int total_array_bytes = N * sizeof(float);
  int half_size = N / 2;
  int half_array_bytes = half_size * sizeof(float);

  //BEGIN: CPU implementation (do not modify)
  float *h_cpu_result = (float*)malloc(N*sizeof(float));
  float *h_input = (float*)malloc(N*sizeof(float));
  //Initialize data on CPU
  int i;
  for (i=0; i<N; i++)
  {
    h_input[i] = 0.1f * i;
  }

  //Execute and time the CPU version
  long long CPU_start_time = start_timer();
  sine_serial(h_input, h_cpu_result);
  long long CPU_time = stop_timer(CPU_start_time, "\nCPU Run Time");
  //END: CPU implementation (do not modify)


  //TODO: Prepare and run your kernel, make sure to copy your results back into h_gpu_result and display your timing results
  // allocating the results array on the host (cpu)
  float *h_gpu_result = (float*)malloc(total_array_bytes);

  // declare 4 pointers (because 2 devices) to memory on the GPU
  float *d_in_1;
  float *d_out_1;
  float *d_out_2;
  float *d_in_2;
 
  // explicitly set which device is being used 
  hipSetDevice(0);
  // insert some timing code now
  long long GPU_memory_allocation_start_time = start_timer(); 
  // now actually allocate GPU memory for input and output
  hipMalloc((void **) &d_in_1, half_array_bytes); 
   
  hipMalloc((void **) &d_out_1, half_array_bytes);

  // using the second device because why not
  hipSetDevice(1);
  hipMalloc((void **) &d_in_2, half_array_bytes);
  hipMalloc((void **) &d_out_2, half_array_bytes);
 
  long long GPU_memory_allocation_time = stop_timer(GPU_memory_allocation_start_time, "\nGPU Memory Allocation"); 

  // time the memory copy to devices
  long long host_to_device_start_time = start_timer(); 
  // the second thing to do would be to copy the input array over into the gpu 2's memory
  hipMemcpy(d_in_2, h_input, half_array_bytes, hipMemcpyHostToDevice); 
 
  hipSetDevice(0); 
  //copy the other half of the input array over onto the first device
  hipMemcpy(d_in_1, &h_input[half_size], half_array_bytes, hipMemcpyHostToDevice);
  long long host_to_device_time = stop_timer(host_to_device_start_time, "GPU Memory Copy to Device");

  // time how long it takes for the kernel to run
  long long kernel_start_time = start_timer(); 
  // now I think I'm ready to launch the kernel on the GPU
  // my original call was faulty since I can't run more than 1024 threads per block!
  
  sine_parallel<<<65535, 1024>>>(d_in_1, d_out_1);
  // also run the kernel on the second device
  hipSetDevice(1);
  sine_parallel<<<65535, 1024>>>(d_in_2, d_out_2);

  // checking to see that there were no errors with the kernel parameters when it got launched
  long long kernel_time = stop_timer(kernel_start_time, "GPU Kernel Run Time");

  // time how long it takes to copy the results on the GPU back onto the CPU
  long long device_to_host_start_time = start_timer(); 
  // now copy the results on GPU 2's memory to CPU memory
  hipMemcpy(h_gpu_result, d_out_2, half_array_bytes, hipMemcpyDeviceToHost); 
  // now copy the results on GPU 1's memory to CPU memory
  hipMemcpy(&h_gpu_result[half_size], d_out_1, half_array_bytes, hipMemcpyDeviceToHost);
  long long device_to_host_time = stop_timer(device_to_host_start_time, "GPU Memory Copy to Host");
  
  // get the total time on the GPU
  long long total_time = stop_timer(GPU_memory_allocation_start_time, "Total GPU Run Time");
  std::cout << "\n";
  // Checking to make sure the CPU and GPU results match - Do not modify
  int errorCount = 0;
  for (i=0; i<N; i++)
  {
    if (abs(h_cpu_result[i]-h_gpu_result[i]) > 1e-6)
      errorCount = errorCount + 1;
  }
  if (errorCount > 0)
    printf("Result comparison failed.\n");
  else
    printf("Result comparison passed.\n");

  // Cleaning up memory
  free(h_input);
  free(h_cpu_result);
  free(h_gpu_result);

  // make sure to free the memory on the GPU too!
  hipFree(d_in_1);
  hipFree(d_out_1);
  hipFree(d_in_2);
  hipFree(d_out_2);

  return 0;
}






